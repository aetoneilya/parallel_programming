#include "hip/hip_runtime.h"
#include "scan.h"

#define MAX_BLOCK_SZ 128
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)

__global__ void gpu_add_block_sums(unsigned int* const d_out,
                                   const unsigned int* const d_in,
                                   unsigned int* const d_block_sums,
                                   const size_t numElems) {
  unsigned int d_block_sum_val = d_block_sums[blockIdx.x];

  unsigned int cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
  if (cpy_idx < numElems) {
    d_out[cpy_idx] = d_in[cpy_idx] + d_block_sum_val;
    if (cpy_idx + blockDim.x < numElems)
      d_out[cpy_idx + blockDim.x] =
          d_in[cpy_idx + blockDim.x] + d_block_sum_val;
  }
}

__global__ void gpu_prescan(unsigned int* const d_out,
                            const unsigned int* const d_in,
                            unsigned int* const d_block_sums,
                            const unsigned int len, const unsigned int shmem_sz,
                            const unsigned int max_elems_per_block) {
  extern __shared__ unsigned int s_out[];

  int thid = threadIdx.x;
  int ai = thid;
  int bi = thid + blockDim.x;

  s_out[thid] = 0;
  s_out[thid + blockDim.x] = 0;
  s_out[thid + blockDim.x + (blockDim.x >> LOG_NUM_BANKS)] = 0;

  __syncthreads();

  unsigned int cpy_idx = max_elems_per_block * blockIdx.x + threadIdx.x;
  if (cpy_idx < len) {
    s_out[ai + CONFLICT_FREE_OFFSET(ai)] = d_in[cpy_idx];
    if (cpy_idx + blockDim.x < len)
      s_out[bi + CONFLICT_FREE_OFFSET(bi)] = d_in[cpy_idx + blockDim.x];
  }

  int offset = 1;
  for (int d = max_elems_per_block >> 1; d > 0; d >>= 1) {
    __syncthreads();

    if (thid < d) {
      int ai = offset * ((thid << 1) + 1) - 1;
      int bi = offset * ((thid << 1) + 2) - 1;
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      s_out[bi] += s_out[ai];
    }
    offset <<= 1;
  }

  if (thid == 0) {
    d_block_sums[blockIdx.x] =
        s_out[max_elems_per_block - 1 +
              CONFLICT_FREE_OFFSET(max_elems_per_block - 1)];
    s_out[max_elems_per_block - 1 +
          CONFLICT_FREE_OFFSET(max_elems_per_block - 1)] = 0;
  }

  for (int d = 1; d < max_elems_per_block; d <<= 1) {
    offset >>= 1;
    __syncthreads();

    if (thid < d) {
      int ai = offset * ((thid << 1) + 1) - 1;
      int bi = offset * ((thid << 1) + 2) - 1;
      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      unsigned int temp = s_out[ai];
      s_out[ai] = s_out[bi];
      s_out[bi] += temp;
    }
  }
  __syncthreads();

  if (cpy_idx < len) {
    d_out[cpy_idx] = s_out[ai + CONFLICT_FREE_OFFSET(ai)];
    if (cpy_idx + blockDim.x < len)
      d_out[cpy_idx + blockDim.x] = s_out[bi + CONFLICT_FREE_OFFSET(bi)];
  }
}

void sum_scan_blelloch(unsigned int* const d_out,
                       const unsigned int* const d_in, const size_t numElems) {
  checkCudaErrors(hipMemset(d_out, 0, numElems * sizeof(unsigned int)));

  unsigned int block_sz = MAX_BLOCK_SZ / 2;
  unsigned int max_elems_per_block = 2 * block_sz;

  unsigned int grid_sz = numElems / max_elems_per_block;
  if (numElems % max_elems_per_block != 0) grid_sz += 1;

  unsigned int shmem_sz =
      max_elems_per_block + ((max_elems_per_block) >> LOG_NUM_BANKS);

  unsigned int* d_block_sums;
  checkCudaErrors(hipMalloc(&d_block_sums, sizeof(unsigned int) * grid_sz));
  checkCudaErrors(hipMemset(d_block_sums, 0, sizeof(unsigned int) * grid_sz));

  gpu_prescan<<<grid_sz, block_sz, sizeof(unsigned int) * shmem_sz>>>(
      d_out, d_in, d_block_sums, numElems, shmem_sz, max_elems_per_block);

  if (grid_sz <= max_elems_per_block) {
    unsigned int* d_dummy_blocks_sums;
    checkCudaErrors(hipMalloc(&d_dummy_blocks_sums, sizeof(unsigned int)));
    checkCudaErrors(hipMemset(d_dummy_blocks_sums, 0, sizeof(unsigned int)));
    gpu_prescan<<<1, block_sz, sizeof(unsigned int) * shmem_sz>>>(
        d_block_sums, d_block_sums, d_dummy_blocks_sums, grid_sz, shmem_sz,
        max_elems_per_block);
    checkCudaErrors(hipFree(d_dummy_blocks_sums));
  } else {
    unsigned int* d_in_block_sums;
    checkCudaErrors(
        hipMalloc(&d_in_block_sums, sizeof(unsigned int) * grid_sz));
    checkCudaErrors(hipMemcpy(d_in_block_sums, d_block_sums,
                               sizeof(unsigned int) * grid_sz,
                               hipMemcpyDeviceToDevice));
    sum_scan_blelloch(d_block_sums, d_in_block_sums, grid_sz);
    checkCudaErrors(hipFree(d_in_block_sums));
  }
  gpu_add_block_sums<<<grid_sz, block_sz>>>(d_out, d_out, d_block_sums,
                                            numElems);

  checkCudaErrors(hipFree(d_block_sums));
}
