#include "hip/hip_runtime.h"
#include "radix.h"

void SerialRadixSort(uint64_t* array, size_t size) {
  uint64_t* buff = new uint64_t[size];

  for (int bit = LOWER_BIT; bit <= UPPER_BIT; bit++) {
    size_t count_zero = 0;

    for (size_t i = 0; i < size; i++) {
      if (!((array[i] >> bit) & 1)) count_zero++;
    }

    size_t zero_ind = 0;
    size_t one_ind = count_zero;

    for (size_t i = 0; i < size; i++) {
      ((array[i] >> bit) & 1) ? buff[one_ind++] = array[i]
                              : buff[zero_ind++] = array[i];
    }

    std::swap(array, buff);
  }

  delete[] buff;
}

__global__ void Count(uint64_t* d_array, size_t arrary_size, uint* d_count,
                      size_t count_size, int segment_size, int bit) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int count_zero = 0;
  int count_one = 0;

  for (int i = 0; i < segment_size; i++) {
    //!
    if ((index * segment_size + i) >= arrary_size) break;

    if ((d_array[index * segment_size + i] >> bit) & 1) {
      count_one++;
    } else {
      count_zero++;
    }
  }

  d_count[index * 2] = count_zero;
  d_count[index * 2 + 1] = count_one;
}

__global__ void Prefix(uint64_t* d_array, size_t arrary_size, uint64_t* d_buff,
                       uint* d_count, size_t count_size, int segment_size,
                       int bit) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  int count_zero = 0;
  int count_one = 0;

  for (int i = 0; i < count_size / 2; i++) {
    count_one += d_count[i * 2];
  }

  for (int i = 0; i < index; i++) {
    count_zero += d_count[i * 2];
    count_one += d_count[i * 2 + 1];
  }

  //   if (index == 0) {
  //     for (int i = 0; i < count_size; i++) {
  //       printf("dc %d\n", d_count[i]);
  //     }
  //   }

  //   printf("index %d | count one %d | count zero %d\n", index, count_one,
  //          count_zero);

  for (int i = 0; i < segment_size; i++) {
    //!
    if ((index * segment_size + i) >= arrary_size) break;

    if ((d_array[index * segment_size + i] >> bit) & 1) {
      d_buff[index * segment_size + i] = count_one++;
    } else {
      d_buff[index * segment_size + i] = count_zero++;
    }
  }

  //   if (index == 0) {
  //     for (int i = 0; i < arrary_size; i++) {
  //       printf("d_arr[%d] = %d\n", i, (int)d_array[i]);
  //     }
  //     for (int i = 0; i < arrary_size; i++) {
  //       printf("d_buf[%d] = %d\n", i, (int)d_buff[i]);
  //     }
  //   }
}

__global__ void MoveByPrefix(uint64_t* d_array, size_t arrary_size,
                             uint64_t* d_buff, uint64_t* d_prefix) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;

  if (index < arrary_size) {
    d_buff[d_prefix[index]] = d_array[index];
  }
}

void ParrallelRadixSort(uint64_t* array, size_t size, int segment_size) {
  uint64_t* d_array;
  uint64_t* d_buff;
  uint64_t* d_prefix;
  hipMalloc(&d_array, sizeof(uint64_t) * size);
  hipMalloc(&d_buff, sizeof(uint64_t) * size);
  hipMalloc(&d_prefix, sizeof(uint64_t) * size);
  hipMemcpy(d_array, (void*)array, sizeof(uint64_t) * size,
             hipMemcpyHostToDevice);
  //   hipMemset(d_buff, 0, sizeof(uint64_t) * size);

  segment_size = THREADS_PER_BLOCK;

  unsigned int blocks = size / segment_size;
  if (size % segment_size != 0) blocks += 1;

  uint* d_count;
  size_t count_size = blocks * 2;
  hipMalloc(&d_count, sizeof(uint) * count_size);

  for (int bit = LOWER_BIT; bit <= UPPER_BIT; bit++) {
    Count<<<blocks, 1>>>(d_array, size, d_count, count_size, segment_size, bit);
    // hipDeviceSynchronize();
    Prefix<<<blocks, 1>>>(d_array, size, d_prefix, d_count, count_size,
                          segment_size, bit);
    // hipDeviceSynchronize();
    MoveByPrefix<<<blocks, THREADS_PER_BLOCK>>>(d_array, size, d_buff,
                                                d_prefix);
    hipDeviceSynchronize();

    std::swap(d_array, d_buff);
    // break;
  }

  hipMemcpy(array, (void*)d_array, sizeof(uint64_t) * size,
             hipMemcpyDeviceToHost);

  //   printf("aaaaaaaaaaaaaaaaaaaaaaaaaaaaaaaa\n");
  //   PrintArray(array, size);

  hipFree(d_array);
  hipFree(d_buff);
  hipFree(d_count);
}
