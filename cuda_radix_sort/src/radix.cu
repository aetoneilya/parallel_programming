#include "hip/hip_runtime.h"
#include "radix.h"

void SerialRadixSort(uint* array, size_t size) {
  uint* buff = new uint[size];

  for (uint bit = LOWER_BIT; bit <= UPPER_BIT; bit++) {
    size_t count_zero = 0;

    for (size_t i = 0; i < size; i++) {
      if (!((array[i] >> bit) & 1)) count_zero++;
    }

    size_t zero_ind = 0;
    size_t one_ind = count_zero;

    for (size_t i = 0; i < size; i++) {
      ((array[i] >> bit) & 1) ? buff[one_ind++] = array[i]
                              : buff[zero_ind++] = array[i];
    }

    std::swap(array, buff);
  }

  delete[] buff;
}

__global__ void gpu_radix_sort_local(unsigned int* d_out_sorted,
                                     unsigned int* d_prefix_sums,
                                     unsigned int* d_block_sums,
                                     unsigned int input_shift_width,
                                     unsigned int* d_in, unsigned int d_in_len,
                                     unsigned int max_elems_per_block) {
  extern __shared__ unsigned int shmem[];
  unsigned int* s_data = shmem;
  unsigned int s_mask_out_len = max_elems_per_block + 1;
  unsigned int* s_mask_out = &s_data[max_elems_per_block];
  unsigned int* s_merged_scan_mask_out = &s_mask_out[s_mask_out_len];
  unsigned int* s_mask_out_sums = &s_merged_scan_mask_out[max_elems_per_block];
  unsigned int* s_scan_mask_out_sums = &s_mask_out_sums[4];

  unsigned int thid = threadIdx.x;

  unsigned int cpy_idx = max_elems_per_block * blockIdx.x + thid;
  if (cpy_idx < d_in_len)
    s_data[thid] = d_in[cpy_idx];
  else
    s_data[thid] = 0;

  __syncthreads();

  unsigned int t_data = s_data[thid];
  unsigned int t_2bit_extract = (t_data >> input_shift_width) & 3;

  for (unsigned int i = 0; i < 4; ++i) {
    s_mask_out[thid] = 0;
    if (thid == 0) s_mask_out[s_mask_out_len - 1] = 0;

    __syncthreads();

    bool val_equals_i = false;
    if (cpy_idx < d_in_len) {
      val_equals_i = t_2bit_extract == i;
      s_mask_out[thid] = val_equals_i;
    }
    __syncthreads();

    int partner = 0;
    unsigned int sum = 0;
    unsigned int max_steps = (unsigned int)log2f(max_elems_per_block);
    for (unsigned int d = 0; d < max_steps; d++) {
      partner = thid - (1 << d);
      if (partner >= 0) {
        sum = s_mask_out[thid] + s_mask_out[partner];
      } else {
        sum = s_mask_out[thid];
      }
      __syncthreads();
      s_mask_out[thid] = sum;
      __syncthreads();
    }

    unsigned int cpy_val = 0;
    cpy_val = s_mask_out[thid];
    __syncthreads();
    s_mask_out[thid + 1] = cpy_val;
    __syncthreads();

    if (thid == 0) {
      s_mask_out[0] = 0;
      unsigned int total_sum = s_mask_out[s_mask_out_len - 1];
      s_mask_out_sums[i] = total_sum;
      d_block_sums[i * gridDim.x + blockIdx.x] = total_sum;
    }
    __syncthreads();

    if (val_equals_i && (cpy_idx < d_in_len)) {
      s_merged_scan_mask_out[thid] = s_mask_out[thid];
    }

    __syncthreads();
  }

  if (thid == 0) {
    unsigned int run_sum = 0;
    for (unsigned int i = 0; i < 4; ++i) {
      s_scan_mask_out_sums[i] = run_sum;
      run_sum += s_mask_out_sums[i];
    }
  }

  __syncthreads();

  if (cpy_idx < d_in_len) {
    unsigned int t_prefix_sum = s_merged_scan_mask_out[thid];
    unsigned int new_pos = t_prefix_sum + s_scan_mask_out_sums[t_2bit_extract];

    __syncthreads();

    s_data[new_pos] = t_data;
    s_merged_scan_mask_out[new_pos] = t_prefix_sum;

    __syncthreads();

    d_prefix_sums[cpy_idx] = s_merged_scan_mask_out[thid];
    d_out_sorted[cpy_idx] = s_data[thid];
  }
}

__global__ void gpu_glbl_shuffle(unsigned int* d_out, unsigned int* d_in,
                                 unsigned int* d_scan_block_sums,
                                 unsigned int* d_prefix_sums,
                                 unsigned int input_shift_width,
                                 unsigned int d_in_len,
                                 unsigned int max_elems_per_block) {
  unsigned int thid = threadIdx.x;
  unsigned int cpy_idx = max_elems_per_block * blockIdx.x + thid;

  if (cpy_idx < d_in_len) {
    unsigned int t_data = d_in[cpy_idx];
    unsigned int t_2bit_extract = (t_data >> input_shift_width) & 3;
    unsigned int t_prefix_sum = d_prefix_sums[cpy_idx];
    unsigned int data_glbl_pos =
        d_scan_block_sums[t_2bit_extract * gridDim.x + blockIdx.x] +
        t_prefix_sum;
    __syncthreads();
    d_out[data_glbl_pos] = t_data;
  }
}

void ParrallelRadixSort(uint* array, size_t size, int segment_size) {
  unsigned int block_sz = segment_size;
  unsigned int max_elems_per_block = block_sz;
  unsigned int grid_sz = size / max_elems_per_block;
  if (size % max_elems_per_block != 0) grid_sz += 1;

  unsigned int* d_out;
  unsigned int* d_in;
  hipMalloc(&d_in, sizeof(uint) * size);
  hipMalloc(&d_out, sizeof(uint) * size);
  hipMemcpy(d_in, (void*)array, sizeof(uint) * size, hipMemcpyHostToDevice);

  unsigned int* d_prefix_sums;
  unsigned int d_prefix_sums_len = size;
  checkCudaErrors(
      hipMalloc(&d_prefix_sums, sizeof(unsigned int) * d_prefix_sums_len));
  checkCudaErrors(
      hipMemset(d_prefix_sums, 0, sizeof(unsigned int) * d_prefix_sums_len));

  unsigned int* d_block_sums;
  unsigned int d_block_sums_len = 4 * grid_sz;
  checkCudaErrors(
      hipMalloc(&d_block_sums, sizeof(unsigned int) * d_block_sums_len));
  checkCudaErrors(
      hipMemset(d_block_sums, 0, sizeof(unsigned int) * d_block_sums_len));

  unsigned int* d_scan_block_sums;
  checkCudaErrors(
      hipMalloc(&d_scan_block_sums, sizeof(unsigned int) * d_block_sums_len));
  checkCudaErrors(hipMemset(d_scan_block_sums, 0,
                             sizeof(unsigned int) * d_block_sums_len));

  unsigned int s_data_len = max_elems_per_block;
  unsigned int s_mask_out_len = max_elems_per_block + 1;
  unsigned int s_merged_scan_mask_out_len = max_elems_per_block;
  unsigned int s_mask_out_sums_len = 4;
  unsigned int s_scan_mask_out_sums_len = 4;
  unsigned int shmem_sz =
      (s_data_len + s_mask_out_len + s_merged_scan_mask_out_len +
       s_mask_out_sums_len + s_scan_mask_out_sums_len) *
      sizeof(unsigned int);

  for (unsigned int shift_width = 0; shift_width <= 30; shift_width += 2) {
    gpu_radix_sort_local<<<grid_sz, block_sz, shmem_sz>>>(
        d_out, d_prefix_sums, d_block_sums, shift_width, d_in, size,
        max_elems_per_block);

    sum_scan_blelloch(d_scan_block_sums, d_block_sums, d_block_sums_len);

    gpu_glbl_shuffle<<<grid_sz, block_sz>>>(d_in, d_out, d_scan_block_sums,
                                            d_prefix_sums, shift_width, size,
                                            max_elems_per_block);
  }

  hipMemcpy(array, (void*)d_in, sizeof(uint) * size, hipMemcpyDeviceToHost);

  checkCudaErrors(hipFree(d_scan_block_sums));
  checkCudaErrors(hipFree(d_block_sums));
  checkCudaErrors(hipFree(d_prefix_sums));
}
